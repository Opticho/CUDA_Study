#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "book.h"

#define N (33 * 1024)


__global__ void add(int* a, int* b, int* c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main(void) {
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;

	// GPU 메모리를 할당한다.
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// CPU로 배열 'a'와 'b'를 채운다.
	for (int i = 0; i < N; ++i)
	{
		a[i] = i;
		b[i] = i * i;
	}

	// 배열 'a'와 'b'를 GPU로 복사한다.
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

	add <<<128, 128>>>(dev_a, dev_b, dev_c);

	// 배열 'c'를 GPU에서 다시 CPU로 복사한다.
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	// 결과를 출력한다.
	for (int i = 0; i < N; ++i)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	//우리가 요청한 작업을 GPU가 수행했는지 확인한다.
	bool success = true;
	for (int i = 0; i < N; ++i)
	{
		if ((a[i] + b[i]) != c[i]) {
			printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}

	if (success) printf("We did it!\n");

	// GPU에 헬당된 메모리를 해제한다.
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}