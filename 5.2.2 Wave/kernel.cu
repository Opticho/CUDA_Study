#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "book.h"
#include "cpu_anim.h"

#include <stdio.h>

#define DIM 1024

struct DataBlock {
	unsigned char* dev_bitmap;
	CPUAnimBitmap* bitmap;
};

// GPU에 할당한 메모리를 해제한다.
void cleanup(DataBlock* d) {
	hipFree(d->dev_bitmap);
}

__global__ void kernel(unsigned char* ptr, int ticks) {
	// threadIdx/blockIdx로 픽셀 위치를 결정한다.
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// 이제 해당 위치의 값을 계산한다.
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

	ptr[offset * 4 + 0] = grey;
	ptr[offset * 4 + 1] = grey;
	ptr[offset * 4 + 2] = grey;
	ptr[offset * 4 + 3] = 255;
}

void generate_frame(DataBlock* d, int ticks) {
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel << <blocks, threads >> > (d->dev_bitmap, ticks);

	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost));
}

int main(void) {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
	
	bitmap.anim_and_exit((void(*)(void*, int))generate_frame, (void(*)(void*))cleanup);
}