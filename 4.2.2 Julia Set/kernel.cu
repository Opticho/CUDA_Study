#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "book.h"
#include "cpu_bitmap.h"

#include <stdio.h>

static const int DIM = 1000;

struct hipComplex {
    float r;
    float i;

    //hipComplex(float a, float b) : r(a), i(b){}

    //float magnitude2(void) { return r * r + i * i; }
    //hipComplex operator*(const hipComplex& a) {
    //    return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    //}
    //hipComplex operator+(const hipComplex& a) {
    //    return hipComplex(r + a.r, i + a.i);
    //}

    // 생성자도 __device__ 키워드를 넣어줘야한다.
    __device__ hipComplex(float a, float b) : r(a), i(b){}

    __device__ float magnitude2(void) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

//int julia_cpu(int x, int y) {
//    const float scale = 1.5f;
//    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
//    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
//
//    hipComplex c(-0.8f, 0.156f);
//    hipComplex a(jx, jy);
//
//    int i = 0;
//    for (i = 0; i < 200; ++i)
//    {
//        a = a * a + c;
//        if (a.magnitude2() > 1000)
//            return 0;
//    }
//
//    return 1;
//}

__device__ int julia(int x, int y) {
    const float scale = 1.5f;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8f, 0.156f);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; ++i) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

//void kernel_cpu(unsigned char* ptr) {
//    for (int y = 0; y < DIM; ++y) {
//        for (int x = 0; x < DIM; ++x) {
//            int offset = x + y * DIM;
//
//            int juliaValue = julia_cpu(x, y);
//            ptr[offset * 4 + 0] = 255 * juliaValue;
//            ptr[offset * 4 + 1] = 0;
//            ptr[offset * 4 + 2] = 0;
//            ptr[offset * 4 + 3] = 255;
//        }
//    }
//}

__global__ void kernel(unsigned char* ptr) {
    // threadIdx/blockIdx로 픽셀 위치를 결정한다.
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // 이제 해당 위치의 값을 계산한다.
    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

int main()
{
    CPUBitmap bitmap(DIM, DIM);

    //unsigned char* ptr = bitmap.get_ptr();
    //kernel_cpu(ptr);

    unsigned char* dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

    dim3 grid(DIM, DIM);
    kernel << <grid, 1 >> > (dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

    bitmap.display_and_exit();

    hipFree(dev_bitmap);

    return 0;
}
