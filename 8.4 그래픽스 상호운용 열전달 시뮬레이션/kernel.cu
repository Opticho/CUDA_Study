#include "hip/hip_runtime.h"
﻿#define GL_GLEXT_PROTOTYPES
#include "GL\glew.h"
#include "GL\glut.h"

#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "hip/hip_runtime.h"
#include ""

#include "book.h"

#include <math.h>

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

static void key_func(unsigned char key, int x, int y);

struct GPUAnimBitmap;

GPUAnimBitmap* static_bitmap;
static GPUAnimBitmap** get_bitmap_ptr() {
	return &static_bitmap;
}

static void draw_func(void) {
	glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glutSwapBuffers();
}

// 갱신 루틴에서 필요로 하는 전역 데이터들
struct DataBlock {
	unsigned char* output_bitmap;
	float* dev_inSrc;
	float* dev_outSrc;
	float* dev_constSrc;
	GPUAnimBitmap* bitmap;
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};

// GLUT 콜백을 위한 정적 함수 사용
static void idle_func(void);

// GPU에 할당한 메모리를 해제한다.
void cleanup(DataBlock* d);

struct GPUAnimBitmap {
	GLuint bufferObj;
	hipGraphicsResource* resource;
	int width, height;
	DataBlock* dataBlock;
	void (*fAnim)(uchar4*, void*, int);
	void (*animExit)(void*);
	void (*clickDrag)(void*, int, int, int, int);
	int dragStartX, dragStartY;

	GPUAnimBitmap(int w, int h, DataBlock* d) {
		width = w;
		height = h;
		dataBlock = d;
		clickDrag = NULL;

		// 우선 하나의 CUDA 디바이스를 찾은 후, 그래픽 상호운용을 설정한다.
		hipDeviceProp_t prop;
		int dev;
		memset(&prop, 0, sizeof(hipDeviceProp_t));
		prop.major = 1;
		prop.minor = 0;
		HANDLE_ERROR(hipChooseDevice(&dev, &prop));

		HANDLE_ERROR(cudaGLSetGLDevice(dev));

		int c = 1;
		char* foo = "name";
		glutInit(&c, &foo);

		glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
		glutInitWindowSize(width, height);
		glutCreateWindow("bitmap");

		glewInit();

		glGenBuffers(1, &bufferObj);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);

		glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * 4, NULL, GL_DYNAMIC_DRAW_ARB);

		HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone));

		//dataBlock = new DataBlock();
		//dataBlock->bitmap = this;
		//hipMalloc((void**)&dataBlock->output_bitmap, width * height * 4);
	}

	~GPUAnimBitmap() {
		cleanup(dataBlock);
	}

	void anim_and_exit(void (*f)(uchar4*, void*, int), void(*e)(void*)) {
		fAnim = f;

		static_bitmap = this;

		// GLUT를 설정하고 주 루프를 시작한다.
		glutKeyboardFunc(key_func);
		glutDisplayFunc(draw_func);
		glutIdleFunc(idle_func);
		glutMainLoop();
	}

	size_t image_size() {
		return width * height * 4;
	}
};

static void key_func(unsigned char key, int x, int y) {
	switch (key) {
	case 27:
		// OpenGL과 CUDA를 정리한다.
		GPUAnimBitmap * bitmap = *(get_bitmap_ptr());
		HANDLE_ERROR(hipGraphicsUnregisterResource(bitmap->resource));
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
		glDeleteBuffers(1, &bitmap->bufferObj);
		exit(0);
	}
}

static void idle_func(void) {
	static int ticks = 1;
	GPUAnimBitmap* bitmap = *(get_bitmap_ptr());
	uchar4* devPtr;
	size_t size;
	HANDLE_ERROR(hipGraphicsMapResources(1, &(bitmap->resource), NULL));
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, bitmap->resource));

	bitmap->fAnim(devPtr, bitmap->dataBlock, ticks++);

	HANDLE_ERROR(hipGraphicsUnmapResources(1, &(bitmap->resource), NULL));

	glutPostRedisplay();
}

// GPU에 할당한 메모리를 해제한다.
void cleanup(DataBlock* d) {
	hipFree(d->output_bitmap);
}

texture<float, 2> texConstSrc;
texture<float, 2> texIn;
texture<float, 2> texOut;

void anim_exit(DataBlock* d) {
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstSrc);

	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);

	HANDLE_ERROR(hipEventDestroy(d->start));
	HANDLE_ERROR(hipEventDestroy(d->stop));
}

__global__ void copy_const_kernel(float* iptr) {
	// threadIdx/blockIdx로 픽셀 위치를 결정한다.
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex2D(texConstSrc, x, y);
	if (c != 0)
		iptr[offset] = c;
}

__global__ void blend_kernel(float* dst, bool dstOut) {
	// threadIdx/blockIdx로 픽셀 위치를 결정한다.
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) left++;
	if (x == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0) top += DIM;
	if (y == DIM - 1) bottom -= DIM;

	float t, l, c, r, b;
	if (dstOut) {
		t = tex2D(texIn, x, y - 1);
		l = tex2D(texIn, x - 1, y);
		c = tex2D(texIn, x, y);
		r = tex2D(texIn, x + 1, y);
		b = tex2D(texIn, x, y + 1);
	}
	else {
		t = tex2D(texOut, x, y - 1);
		l = tex2D(texOut, x - 1, y);
		c = tex2D(texOut, x, y);
		r = tex2D(texOut, x + 1, y);
		b = tex2D(texOut, x, y + 1);
	}
	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

void anim_gpu(uchar4* outputBitmap, DataBlock* d, int ticks) {
	HANDLE_ERROR(hipEventRecord(d->start, 0));
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);

	// 텍스처가 전역으로 선언되고 바인딩되었으므로,
	// 루프가 반복될 때마다 플래그를 이용하여 입력과 출력을 결정한다.
	volatile bool dstOut = true;
	for (int i = 0; i < 90; ++i) {
		float* in, * out;
		if (dstOut) {
			in = d->dev_inSrc;
			out = d->dev_outSrc;
		}
		else {
			out = d->dev_inSrc;
			in = d->dev_outSrc;
		}
		copy_const_kernel << <blocks, threads >> > (in);
		blend_kernel << <blocks, threads >> > (out, dstOut);
		dstOut = !dstOut;
	}

	float_to_color << <blocks, threads >> > (outputBitmap, d->dev_inSrc);

	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));

	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average Timeper frame: %3.1f ms\n", d->totalTime / d->frames);
}

int main(void) {
	DataBlock data;
	GPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;
	HANDLE_ERROR(hipEventCreate(&data.start));
	HANDLE_ERROR(hipEventCreate(&data.stop));

	int imageSize = bitmap.image_size();
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	HANDLE_ERROR(hipMalloc((void**)&data.output_bitmap, bitmap.image_size()));

	size_t offset;

	// float의 크기는 4개의 char와 같다고 가정한다.(즉, rgba)
	HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));

	const textureReference* refConstSrc = new textureReference();
	const textureReference* refIn = new textureReference();
	const textureReference* refOut = new textureReference();

	hipGetTextureReference(&refConstSrc, HIP_SYMBOL(&texConstSrc));
	hipGetTextureReference(&refIn, HIP_SYMBOL(&texIn));
	hipGetTextureReference(&refOut, HIP_SYMBOL(&texOut));

	hipBindTexture2D(&offset, refConstSrc, data.dev_constSrc, &desc, DIM, DIM, sizeof(float) * DIM);
	hipBindTexture2D(&offset, refIn, data.dev_inSrc, &desc, DIM, DIM, sizeof(float) * DIM);
	hipBindTexture2D(&offset, refOut, data.dev_outSrc, &desc, DIM, DIM, sizeof(float) * DIM);


	float* temp = (float*)malloc(bitmap.image_size());
	for (int i = 0; i < DIM * DIM; ++i) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
			temp[i] = MAX_TEMP;
	}

	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for (int y = 800; y < 900; ++y) {
		for (int x = 400; x < 500; ++x) {
			temp[x + y * DIM] = MIN_TEMP;
		}
	}

	HANDLE_ERROR(hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));

	for (int y = 800; y < DIM; ++y) {
		for (int x = 0; x < 200; ++x) {
			temp[x + y * DIM] = MAX_TEMP;
		}
	}

	HANDLE_ERROR(hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));

	free(temp);

	bitmap.anim_and_exit((void(*)(uchar4*, void*, int))anim_gpu, (void (*)(void*))anim_exit);
}