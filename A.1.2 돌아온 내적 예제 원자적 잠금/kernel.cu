#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "book.h"
#include "lock.h"

#include <stdio.h>

#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(Lock lock, float* a, float* b, float* c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// 캐시 값들을 설정한다.
	cache[cacheIndex] = temp;

	__syncthreads();

	// 다음 코드 때문에 리덕션을 위해서는 threadsPerBlock은 2의 멱수여야 한다.
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
		{
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
	{
		lock.lock();
		*c += cache[0];
		lock.unlock();
	}
}

int main(void) {
	Lock lock;

	float* a, * b, c = 0;
	float* dev_a, * dev_b, * dev_c;

	// CPU 측의 메모리를 할당한다.
	a = new float[N];
	b = new float[N];

	// GPU 메모리를 할당한다.
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(float)));

	// 호스트 메모리에 데이터를 채운다.
	for (int i = 0; i < N; ++i) {
		a[i] = i;
		b[i] = i * 2;
	}

	// 배열 'a'와 'b'를 GPU로 복사한다.
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, &c, sizeof(float), hipMemcpyHostToDevice));

	dot << <blocksPerGrid, threadsPerBlock >> > (lock, dev_a, dev_b, dev_c);

	// 배열 'C'를 GPU에서 CPU로 복사한다.
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(float), hipMemcpyDeviceToHost));

#define sum_squares(x) (x*(x+1)*(2*x+1)/6)
	printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares((float)(N - 1)));

	// GPU에 헬당된 메모리를 해제한다.
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	// CPU쪽에서 메모리를 해제한다.
	delete[] a;
	delete[] b;
}