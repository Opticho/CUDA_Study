#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>


hipError_t addition8WithCuda(unsigned char* input1, unsigned char* input2, unsigned char* output, int xSize, int ySize);

int verify(unsigned char* input, unsigned char* output, int xSize, int ySize);

int verify(unsigned char* GoldInput, unsigned char* CudaInput, int xSize, int ySize) {
	for (int i = 0; i < xSize * ySize; i++) {
		if (GoldInput[i] != CudaInput[i]) {
			printf("GoldInput[%d] = %d CInput[%d]=%d \n", i, GoldInput[i], i, CudaInput[i]);
			return(1);
		}
	}
	return(0);
}


void addition8WithC(unsigned char* in1, unsigned char* in2, unsigned char* out, size_t x_size, size_t y_size) {
	size_t x, y;
	for (y = 0; y < y_size; y += 1) {
		for (x = 0; x < x_size; x += 1) {
			size_t pos = y * x_size + x;
			*(out + pos) = *(in1 + pos) / 2 + *(in2 + pos) / 2;
		}
	}
}


__global__ void kernelAddition8(unsigned char* input1, unsigned char* input2, unsigned char* output, int size) {
	int xWidth = blockDim.x * gridDim.x;
	int xLoc = (blockIdx.x * blockDim.x + threadIdx.x);
	int yLoc = blockIdx.y * blockDim.y + threadIdx.y;


	int index = xLoc + yLoc * xWidth;

	if (index < (size))
		output[index] = input1[index] / 2 + input2[index] / 2;
}

int main() {
	unsigned char* in1, *in2, * CudaOutput, * GoldOutput;
	int xSize, ySize;

	xSize = 512;
	ySize = 512;
	in1 = new unsigned char[xSize * ySize];
	in2 = new unsigned char[xSize * ySize];
	CudaOutput = new unsigned char[xSize * ySize];
	GoldOutput = new unsigned char[xSize * ySize];
	int i, j;
	printf("xSize=%d ySize=%d \n", xSize, ySize);

	FILE* fp;

	//fp = fopen("barbara_gray.raw", "rb");

	//fread(input, xSize, ySize, fp);

	for (int i = 0; i < ySize; i++)
		for (int j = 0; j < xSize; j++) {
			in1[i * xSize + j] = (i * j) % 255;
			in2[i * xSize + j] = (i * j) % 100;
		}

	addition8WithC(in1, in2, GoldOutput, xSize, ySize);
	// Add vectors in parallel.
	hipError_t cudaStatus = addition8WithCuda(in1, in2, CudaOutput, xSize, ySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "invert8WithCuda failed!");
		return 1;
	}

	int error = verify(GoldOutput, CudaOutput, xSize, ySize);

	if (error != 0)
		printf("Verify Failed \n");
	else
		printf("Verify Successful \n");

	fp = fopen("COutput.raw", "wb");
	fwrite(GoldOutput, xSize, ySize, fp);
	fclose(fp);

	fp = fopen("CudaOutput.raw", "wb");
	fwrite(CudaOutput, xSize, ySize, fp);
	fclose(fp);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	delete[] CudaOutput;
	delete[] GoldOutput;
	delete[] in1;
	delete[] in2;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addition8WithCuda(unsigned char* in1, unsigned char* in2, unsigned char* output, int xSize, int ySize) {
	unsigned char* dev_input1 = 0;
	unsigned char* dev_input2 = 0;
	unsigned char* dev_output = 0;

	//	cudaProfilerInitialize();
	unsigned int xysize = xSize * ySize;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.

	hipDeviceProp_t prop;
	int count;

	dim3 blocks, threads;

	threads.x = 512;
	threads.y = 1;
	//512x512 : along X 512/8 = 64 thread blocks Alon gY 64 blocks
	blocks.x = (xSize + threads.x - 1) / (threads.x); //1
	blocks.y = (ySize + threads.y - 1) / (threads.y); //512
	printf("blocks.x = %d blocks.y=%d \n", blocks.x, blocks.y);
	printf("threads.x = %d threads.y=%d \n", threads.x, threads.y);


	hipGetDeviceCount(&count);
	printf("Count =  %d\n", count);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	hipEventRecord(start, 0);
	// Allocate GPU buffers for two input     .
	cudaStatus = hipMalloc((void**)&dev_input1, xysize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for two input     .
	cudaStatus = hipMalloc((void**)&dev_input2, xysize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_output, xysize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input1, in1, xysize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input2, in2, xysize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipProfilerStart();

	// Launch a kernel on the GPU with one thread for each element.
	kernelAddition8 __dim__(blocks, threads) (dev_input1, dev_input2, dev_output, xysize);

	hipProfilerStop();
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching invert8Kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_output, xysize * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);


	float cudaElapsedTime;
	hipEventElapsedTime(&cudaElapsedTime, start, stop);
	printf("Time for execution = %3.1f ms \n", cudaElapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

Error:
	hipFree(dev_input1);
	hipFree(dev_input2);
	hipFree(dev_output);

	return cudaStatus;
}


