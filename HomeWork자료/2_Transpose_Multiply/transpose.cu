#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>


hipError_t transpose8WithCuda(unsigned char* input, unsigned char* output, int xSize, int ySize);

int verify(unsigned char* input, unsigned char* output, int xSize, int ySize);

int verify(unsigned char* GoldInput, unsigned char* CudaInput, int xSize, int ySize) {
	for (int i = 0; i < xSize * ySize; i++) {
		if (GoldInput[i] != CudaInput[i]) {
			printf("GoldInput[%d] = %d CInput[%d]=%d \n", i, GoldInput[i], i, CudaInput[i]);
			return(1);
		}
	}
	return(0);
}


void transpose8WithC(unsigned char* in, unsigned char* out, int x_size, int y_size) {
	size_t x, y;
	for (y = 0; y < y_size; y += 1) {
		for (x = 0; x < x_size; x += 1) {
			size_t in_pos = y * x_size + x;
			size_t out_pos = x * y_size + y;
			*(out + out_pos) = *(in + in_pos);
		}
	}
}


__global__ void kernelTranspose8(unsigned char* input, unsigned char* output, int size) {
	int xWidth = blockDim.x * gridDim.x;
	int xLoc = (blockIdx.x * blockDim.x + threadIdx.x);
	int yLoc = blockIdx.y * blockDim.y + threadIdx.y;

	int index = xLoc + yLoc * xWidth;

	if (index < (size)) {
		output[yLoc * xWidth + xLoc] = input[index];
	}
}

int main() {
	unsigned char* input, * CudaOutput, * GoldOutput;
	int xSize, ySize;

	xSize = 512;
	ySize = 512;
	input = new unsigned char[xSize * ySize];
	CudaOutput = new unsigned char[xSize * ySize];
	GoldOutput = new unsigned char[xSize * ySize];
	int i, j;
	printf("xSize=%d ySize=%d \n", xSize, ySize);

	FILE* fp;

	//fp = fopen("barbara_gray.raw", "rb");

	//fread(input, xSize, ySize, fp);

	for (int i = 0; i < ySize; i++)
		for (int j = 0; j < xSize; j++)
			input[i * xSize + j] = (i * j) % 255;

	transpose8WithC(input, GoldOutput, xSize, ySize);
	// Add vectors in parallel.
	hipError_t cudaStatus = transpose8WithCuda(input, CudaOutput, xSize, ySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "invert8WithCuda failed!");
		return 1;
	}

	int error = verify(GoldOutput, CudaOutput, xSize, ySize);

	if (error != 0)
		printf("Verify Failed \n");
	else
		printf("Verify Successful \n");

	fp = fopen("COutput.raw", "wb");
	fwrite(GoldOutput, xSize, ySize, fp);
	fclose(fp);

	fp = fopen("CudaOutput.raw", "wb");
	fwrite(CudaOutput, xSize, ySize, fp);
	fclose(fp);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	delete[] CudaOutput;
	delete[] GoldOutput;
	delete[] input;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t transpose8WithCuda(unsigned char* input, unsigned char* output, int xSize, int ySize) {
	unsigned char* dev_input = 0;
	unsigned char* dev_output = 0;

	//	cudaProfilerInitialize();
	unsigned int xysize = xSize * ySize;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.

	hipDeviceProp_t prop;
	int count;

	dim3 blocks, threads;

	threads.x = 512;
	threads.y = 1;
	//512x512 : along X 512/8 = 64 thread blocks Alon gY 64 blocks
	blocks.x = (xSize + threads.x - 1) / (threads.x); //1
	blocks.y = (ySize + threads.y - 1) / (threads.y); //512
	printf("blocks.x = %d blocks.y=%d \n", blocks.x, blocks.y);
	printf("threads.x = %d threads.y=%d \n", threads.x, threads.y);


	hipGetDeviceCount(&count);
	printf("Count =  %d\n", count);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	hipEventRecord(start, 0);
	// Allocate GPU buffers for two input     .
	cudaStatus = hipMalloc((void**)&dev_input, xysize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_output, xysize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, input, xysize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipProfilerStart();

	// Launch a kernel on the GPU with one thread for each element.
	kernelTranspose8 __dim__(blocks, threads) (dev_input, dev_output, xysize);

	hipProfilerStop();
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching invert8Kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_output, xysize * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);


	float cudaElapsedTime;
	hipEventElapsedTime(&cudaElapsedTime, start, stop);
	printf("Time for execution = %3.1f ms \n", cudaElapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

Error:
	hipFree(dev_input);
	hipFree(dev_output);

	return cudaStatus;
}


