#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "book.h"

#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

float malloc_test(int size);
float cuda_host_alloc_test(int size);

int main(void) {
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
	if (prop.canMapHostMemory != 1) {
		printf("Device cannot map memory.\n");
		return 0;
	}

	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

	float elapsedTime = malloc_test(N);
	printf("Time using hipMalloc: %3.1f ms\n", elapsedTime);

	elapsedTime = cuda_host_alloc_test(N);
	printf("Time using hipHostAlloc: %3.1f ms\n", elapsedTime);
}

__global__ void dot(int size, float* a, float* b, float* c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	float temp = 0;
	while (tid < size) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// 캐시 값들을 설정한다.
	cache[cacheIndex] = temp;

	// 이 블록의 스레드들을 동기화한다.
	__syncthreads();

	// 다음 코드 때문에 리덕션을 위해서는 threadsPerBlock은 2의 멱수어야 한다.
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

float malloc_test(int size) {
	hipEvent_t start, stop;
	float* a, * b, c, * partial_c;
	float* dev_a, * dev_b, * dev_partial_c;
	float elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	// CPU쪽에서 메모리를 할당한다.
	a = (float*)malloc(size * sizeof(float));
	b = (float*)malloc(size * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

	// GPU 메모리를 할당한다.
	HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, size * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float)));

	// 호스트 메모리에 데이터를 채운다.
	for (int i = 0; i < size; ++i) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipEventRecord(start, 0));
	// 배열'a'와 'b'를 GPU로 복사한다.
	HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));

	dot << <blocksPerGrid, threadsPerBlock >> > (size, dev_a, dev_b, dev_partial_c);

	// 배열 'c'를 GPU에서 다시 CPU로 복사한다.
	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	// CPU쪽에서 마무리를 짓는다.
	c = 0;
	for (int i = 0; i < blocksPerGrid; ++i) {
		c += partial_c[i];
	}

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_partial_c));

	// CPU쪽에서 메모리를 해제한다.
	free(a);
	free(b);
	free(partial_c);

	// 이벤트를 해제한다.
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	printf("Value calculated: %d\n", c);

	return elapsedTime;
}

float cuda_host_alloc_test(int size) {
	hipEvent_t start, stop;
	float* a, * b, c, * partial_c;
	float* dev_a, * dev_b, * dev_partial_c;
	float elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	// CPU쪽에서 메모리를 할당한다.
	HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&partial_c, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));

	// 호스트 메모리에 데이터를 채운다.
	for (int i = 0; i < size; ++i) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

	HANDLE_ERROR(hipEventRecord(start, 0));

	dot << <blocksPerGrid, threadsPerBlock >> > (size, dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipDeviceSynchronize());

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	// CPU쪽에서 마무리를 짓는다.
	c = 0;
	for (int i = 0; i < blocksPerGrid; ++i) {
		c += partial_c[i];
	}

	HANDLE_ERROR(hipHostFree(a));
	HANDLE_ERROR(hipHostFree(b));
	HANDLE_ERROR(hipHostFree(partial_c));

	// 이벤트를 해제한다.
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	printf("Value calculated: %d\n", c);

	return elapsedTime;
}